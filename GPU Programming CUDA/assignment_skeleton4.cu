#include "hip/hip_runtime.h"
%%cuda --name week4_Programming_Assignment.cu
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

void print_array(float *A, int N){
    for(int i=0;i<N;i++)
        printf("%.2f ",A[i]);
    printf("\n");
}
//--------------------------------------------------------------------------------------------------------------
__global__ void
process_kernel1(float *input1, float *input2, float *output, int datasize)
{
    int numElements = datasize / sizeof(float);

    //Write code for i
    int blockNum=blockIdx.z*(gridDim.x * gridDim.y)+blockIdx.y * gridDim.x+blockIdx.x;
    int threadNum=threadIdx.z*(blockDim.x* blockDim.y)+threadIdx.y*blockDim.x+threadIdx.x;
    int i= blockNum*blockDim.x*blockDim.y*blockDim.z+threadNum;

    if (i < numElements){
        output[i] = sinf(input1[i]) + cosf(input2[i]);
    }
}


__global__ void
process_kernel2(float *input, float *output, int datasize)
{
    int numElements = datasize / sizeof(float);

     //Write code for i
    int blockNum=blockIdx.z*(gridDim.x * gridDim.y)+blockIdx.y * gridDim.x+blockIdx.x;
    int threadNum=threadIdx.z*(blockDim.x* blockDim.y)+threadIdx.y*blockDim.x+threadIdx.x;
    int i= blockNum*blockDim.x*blockDim.y*blockDim.z+threadNum;
    
    if (i < numElements)
    {
        output[i] = logf(input[i]);
    }
}


__global__ void
process_kernel3(float *input, float *output, int datasize)
{
    int numElements = datasize / sizeof(float);

    //Write code for i
    int blockNum=blockIdx.z*(gridDim.x * gridDim.y)+blockIdx.y * gridDim.x+blockIdx.x;
    int threadNum=threadIdx.z*(blockDim.x* blockDim.y)+threadIdx.y*blockDim.x+threadIdx.x;
    int i= blockNum*blockDim.x*blockDim.y*blockDim.z+threadNum;
    
    if (i < numElements)
    {
        output[i] = sqrtf(input[i]);
    }
}

//------------------------------------------------------------------------------------------------------------------------------------------
int main(void){
    hipError_t err = hipSuccess;

    int numElements = 16384;
    size_t size = numElements * sizeof(float);

    float *h_input1 = (float *)malloc(size);
    float *h_input2 = (float *)malloc(size);
    float *h_output1 = (float *)malloc(size);
    float *h_output2 = (float *)malloc(size);
    float *h_output3 = (float *)malloc(size);

    if (h_input1 == NULL || h_input2 == NULL || h_output1 == NULL || h_output2 == NULL || h_output3 == NULL){
        fprintf(stderr, "Failed to allocate host vectors!\n");
        exit(EXIT_FAILURE);
    }
   
    for (int i = 0; i < numElements; ++i){
        scanf("%f",&h_input1[i]);
    }
    for (int i = 0; i < numElements; ++i){
        scanf("%f",&h_input2[i]);
    }
    
    float *d_input1 = NULL;
    err = hipMalloc((void **)&d_input1, size);

    if (err != hipSuccess){
        fprintf(stderr, "Failed to allocate device vector d_input1 (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    float *d_input2 = NULL;
    err = hipMalloc((void **)&d_input2, size);

    if (err != hipSuccess){
        fprintf(stderr, "Failed to allocate device vector d_input2 (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    float *d_output1 = NULL;
    err = hipMalloc((void **)&d_output1, size);

    if (err != hipSuccess){
        fprintf(stderr, "Failed to allocate device vector h_output1 (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    float *d_output2 = NULL;
    err = hipMalloc((void **)&d_output2, size);

    if (err != hipSuccess){
        fprintf(stderr, "Failed to allocate device vector h_output2 (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    float *d_output3 = NULL;
    err = hipMalloc((void **)&d_output3, size);

    if (err != hipSuccess){
        fprintf(stderr, "Failed to allocate device vector h_output3 (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // printf("Copy input data from the host memory to the CUDA device\n");
    err = hipMemcpy(d_input1, h_input1, size, hipMemcpyHostToDevice);
    if (err != hipSuccess){
        fprintf(stderr, "Failed to copy vector h_input1 from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(d_input2, h_input2, size, hipMemcpyHostToDevice);
    if (err != hipSuccess){
        fprintf(stderr, "Failed to copy vector h_input2 from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

  //Complete Code for launching process_kernel1 -------------------------------------------------------------------------------------
    process_kernel1<<<dim3(4,2,2),dim3(32,32,1)>>>(d_input1, d_input2, d_output1, size);

    err = hipGetLastError();

    if (err != hipSuccess){
        fprintf(stderr, "Failed to launch process_kernel1 kernel (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    //Complete Code for launching process_kernel2 ------------------------------------------------------------------------------------
    process_kernel2<<<dim3(2,8,1),dim3(8,8,16)>>>(d_output1,d_output2, size);

    err = hipGetLastError();

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to launch process_kernel2 kernel (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    //Complete Code for launching process_kernel3 -------------------------------------------------------------------------------------
    process_kernel3<<<dim3(16,1,1),dim3(128,8,1)>>>(d_output2, d_output3, size);

    err = hipGetLastError();

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to launch process_kernel3 kernel (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

//--------------------------------------------------------------------------------------------------------------------------------------------
    
    // printf("Copy output data from the CUDA device to the host memory\n");
    err = hipMemcpy(h_output1, d_output1, size, hipMemcpyDeviceToHost);
    if (err != hipSuccess){
        fprintf(stderr, "Failed to copy vector d_output1 from device to host (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(h_output2, d_output2, size, hipMemcpyDeviceToHost);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector d_output2 from device to host (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(h_output3, d_output3, size, hipMemcpyDeviceToHost);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector d_output3 from device to host (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Verify that the result vectors are as expected

    for (int i = 0; i < numElements; ++i)
    {
        if (fabs(sinf(h_input1[i]) + cosf(h_input2[i]) - h_output1[i]) > 1e-5)
        {
            fprintf(stderr, "Result verification for h_output1 failed at element %d!\n", i);
            exit(EXIT_FAILURE);
        }
    }

    for (int i = 0; i < numElements; ++i)
    {
        if (fabs(logf(h_output1[i]) - h_output2[i]) > 1e-5)
        {
            fprintf(stderr, "Result verification for h_output2 failed at element %d!\n", i);
            exit(EXIT_FAILURE);
        }
    }

    for (int i = 0; i < numElements; ++i)
    {
        if (fabs(sqrtf(h_output2[i]) - h_output3[i]) > 1e-5)
        {
            fprintf(stderr, "Result verification for h_output3 failed at element %d!\n", i);
            exit(EXIT_FAILURE);
        }
    }

    print_array(h_output3,numElements);
  
    err = hipFree(d_input1);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector d_input1 (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipFree(d_input2);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector d_input2 (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipFree(d_output1);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector d_output1 (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipFree(d_output2);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector d_output2 (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipFree(d_output3);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector d_output3 (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    free(h_input1);
    free(h_input2);
    free(h_output1);
    free(h_output2);
    free(h_output3);

    err = hipDeviceReset();
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to deinitialize the device! error=%s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }  
    return 0;
}