#include <stdio.h>
#include <hip/hip_runtime.h>


#define LOG_INPUT if(0)
#define LOG_OUTPUT if(1)
#define LOG if(0)


__global__ void hadamard(float *A, float *B, float *C, int M, int N)
{
    // Complete the kernel code snippet
    int j=blockIdx.x * blockDim.x + threadIdx.x;
    int i=blockIdx.y * blockDim.y + threadIdx.y;
    int t=i*N+j;
    
    if(t<M*N){
        C[t]=A[t]*B[t];
    }
}

/**
 * Host main routine
 */
void print_matrix(float *A,int m,int n)
{
    for(int i =0;i<m;i++)
    {
        for(int j=0;j<n;j++)
            printf("%.2f ",A[i*n+j]);
        printf("\n");
    }

}

int main(void)
{
    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;

    // Print the vector length to be used, and compute its size
    
    int t; //number of test cases
    scanf("%d",&t);
    while(t--)
    {
        int m,n;
        scanf("%d %d",&m,&n);
        size_t size = m*n * sizeof(float);
        LOG printf("[Hadamard product of two matrices ]\n");

        // Allocate the host input vector A
        float *h_A=(float *)malloc(size);
        // Allocate the host input vector B
        float *h_B=(float *)malloc(size);
        // Allocate the host output vector C
        float *h_C=(float *)malloc(size);

        // Verify that allocations succeeded
        if (h_A == NULL || h_B == NULL || h_C == NULL)
        {
            fprintf(stderr, "Failed to allocate host vectors!\n");
            exit(EXIT_FAILURE);
        }

        // Initialize the host input vectors
        
        for (int i = 0; i < n*m; ++i)
        {
            scanf("%f",&h_A[i]);
            scanf("%f",&h_B[i]);

        }
        
        // Allocate the device input vector A
        float *d_A = NULL;
        err=hipMalloc (( void **) &d_A , size );

        // Allocate the device input vector B
        float *d_B = NULL;
        err=hipMalloc (( void **) &d_B , size );
     
        // Allocate the device output vector C
        float *d_C = NULL;
        err=hipMalloc (( void **) &d_C , size );
     
        // Copy the host input vectors A and B in host memory to the device input vectors in
        // device memory
        err = hipMemcpy (d_A , h_A , size , hipMemcpyHostToDevice );
        err = hipMemcpy (d_B , h_B , size , hipMemcpyHostToDevice );
        
        // initialize blocksPerGrid and threads Per Block
        int threadsPerBlock = 256;
        int blocksPerGrid =(m*n)/256;

        hadamard<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, m, n);
        err = hipGetLastError();

        if (err != hipSuccess)
        {
            fprintf(stderr, "Failed to launch vectorAdd kernel (error code %s)!\n", hipGetErrorString(err));
            exit(EXIT_FAILURE);
        }

      
        // Copy the device result vector in device memory to the host result vector
        // in host memory.
        err = hipMemcpy (h_C , d_C , size , hipMemcpyDeviceToHost );
        
        // Verify that the result vector is correct
        for (int i = 0; i < n*m; ++i)
        {
            if (fabs(h_A[i] * h_B[i] - h_C[i]) > 1e-5)
            {
                fprintf(stderr, "Result verification failed at element %d!\n", i);
                fprintf(stderr, "Ideal:%.2f  calculated: %.2f\n",h_A[i] * h_B[i],h_C[i] );
                exit(EXIT_FAILURE);
            }
        }
        LOG printf("Test PASSED\n");

        // Free device global memory
        hipFree(d_A);hipFree(d_B);hipFree(d_C);

        // Free host memory
        free(h_A);free(h_B);
        
        err = hipDeviceReset();

        if (err != hipSuccess)
        {
            fprintf(stderr, "Failed to deinitialize the device! error=%s\n", hipGetErrorString(err));
            exit(EXIT_FAILURE);
        }
        print_matrix(h_C,m,n);
        free(h_C);
        LOG printf("Done\n");
    }
    return 0;
}