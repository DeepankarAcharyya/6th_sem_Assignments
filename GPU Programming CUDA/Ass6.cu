#include <stdio.h>
#include <hip/hip_runtime.h>


__global__ void convolution(float *mat_input1, float *mat_conv_input1, float *mat_output1, int mat_dim);

__global__ void
convolution(float *mat_input1, float *mat_conv_input1, float *mat_output1, int mat_dim)
{
    // Note this has to be a 1D kernel implementation. Check host program for corresponding launch parameters. 
    
    int x1=blockIdx.x * blockDim.x + threadIdx.x;
    int y1=blockIdx.y * blockDim.y + threadIdx.y;
    int i=y1*mat_dim+x1;
    int   j = i/mat_dim;
    int   k = i%mat_dim;

    float avg_val = 0;

    for(int r=0;r<=2;r++){
          for(int c=0;c<=2;c++){
                    if((j-1+r)>=0 && (j-1+r)<mat_dim && (k-1+c)>=0 && (k-1+c)<mat_dim){
                        avg_val += (mat_input1[mat_dim*(j-1+r)+(k-1+c)]*mat_conv_input1[3*r + c]);
                    }
                }
            }
            mat_output1[mat_dim*j + k] = avg_val;
}

void print_matrix(float *A,int m,int n)
{
    for(int i =0;i<m;i++)
    {
        for(int j=0;j<n;j++)
            printf("%.2f ",A[i*n+j]);
        printf("\n");
    }

}

//------------------------------------------------------------------------------------------------------------------------------
int main(void){
    hipError_t err = hipSuccess;

    int i, j, k;
    int t;
    scanf("%d",&t);
    while(t--){
        int mat_dim;
        //float conv_total=0.0;
        scanf("%d",&mat_dim);
        int mat_num_eles = mat_dim*mat_dim;
        size_t mat_size = mat_num_eles*sizeof(float);
        
        float *h_mat_input1 = (float *)malloc(mat_num_eles*sizeof(float));
        float *h_mat_output1 = (float *)malloc(mat_num_eles*sizeof(float));
        float *h_mat_output2 = (float *)malloc(mat_num_eles*sizeof(float));

        int mat_conv_dim = 3;
        int mat_conv_num_eles = mat_conv_dim*mat_conv_dim;
        size_t mat_conv_size = mat_conv_num_eles*sizeof(float);
        float *h_mat_conv_input = (float *)malloc(mat_conv_num_eles*sizeof(float));

        if (h_mat_input1 == NULL || h_mat_output1 == NULL || h_mat_output2 == NULL || h_mat_conv_input == NULL){
            fprintf(stderr, "Failed to allocate host vectors!\n");
            exit(EXIT_FAILURE);
        }
//--------------------storing the elements--------------------------------
        for(i=0;i<mat_num_eles;i++){
            j = i/mat_dim;
            k = i%mat_dim;

            scanf("%f",&h_mat_input1[mat_dim*j + k]); // = (float)(rand()%10);
        }

        for(i=0;i<mat_conv_num_eles;i++)
        {
            j = i/mat_conv_dim;
            k = i%mat_conv_dim;
            h_mat_conv_input[mat_conv_dim*j + k] = 1.0/9.0;
            
        }

//------------------------------------------------------------------------
        
        //Code for allocating device buffers
        float *d_mat_input1= NULL;
        err=hipMalloc (( void **) &d_mat_input1 , mat_size );

        float *d_mat_conv_input= NULL;
        err=hipMalloc (( void **) &d_mat_conv_input , mat_conv_size );

        float *d_mat_output1=NULL;
        err=hipMalloc (( void **) &d_mat_output1 , mat_size );
//------------------------------------------------------------------------
        
        //Code for copying data buffers to device
        err = hipMemcpy (d_mat_input1 , h_mat_input1 , mat_size , hipMemcpyHostToDevice );
        err = hipMemcpy (d_mat_conv_input , h_mat_conv_input , mat_conv_size , hipMemcpyHostToDevice );

//------------------------------------------------------------------------        

        //Code for launching the kernel

        // Hint we have fixed the threads per block for this problem. Note it's a 1D kernel. Design your implementation accordingly
        
        int mat_block_dim = 16;
        int mat_grid_dim = ceil(mat_num_eles/float(mat_block_dim));
        convolution<<<mat_grid_dim, mat_block_dim>>>(d_mat_input1, d_mat_conv_input, d_mat_output1, mat_dim);
       
        err = hipGetLastError();
        if (err != hipSuccess)
        {
            fprintf(stderr, "Failed to launch process_kernel2 kernel (error code %s)!\n", hipGetErrorString(err));
            exit(EXIT_FAILURE);
        }

        //Code for getting back results to host
        err = hipMemcpy (h_mat_output1 , d_mat_output1 , mat_size , hipMemcpyDeviceToHost );
       
//-----------------------------------------------------------------------------------------------------------------
        // Do operations serially
        
        for(i=0;i<mat_num_eles;i++)
        {
            j = i/mat_dim;
            k = i%mat_dim;

            float avg_val = 0;

            for(int r=0;r<=2;r++)
            {
                for(int c=0;c<=2;c++)
                {
                    if((j-1+r)>=0 && (j-1+r)<mat_dim && (k-1+c)>=0 && (k-1+c)<mat_dim)
                    {
                        avg_val += (h_mat_input1[mat_dim*(j-1+r)+(k-1+c)]*h_mat_conv_input[3*r + c]);
                    }
                }
            }

            h_mat_output2[mat_dim*j + k] = avg_val;
        }

        // Code to Verify that the output matrices are as expected
        
      for (int i = 0; i < mat_num_eles; i++)
        {
            if (fabs(h_mat_output1[i]- h_mat_output2[i]) > 1e-5)
            {
                fprintf(stderr, "Result verification failed at element %d!\n", i);
                fprintf(stderr, "Ideal:%.2f  calculated: %.2f\n",h_mat_output2[i],h_mat_output1[i] );
                exit(EXIT_FAILURE);
            }
        }
        
        
        err = hipFree(d_mat_input1);
        if (err != hipSuccess)
        {
            fprintf(stderr, "Failed to free device vector d_mat_input1 (error code %s)!\n", hipGetErrorString(err));
            exit(EXIT_FAILURE);
        }
        err = hipFree(d_mat_conv_input);
        if (err != hipSuccess)
        {
            fprintf(stderr, "Failed to free device vector d_mat_conv_input (error code %s)!\n", hipGetErrorString(err));
            exit(EXIT_FAILURE);
        }
        err = hipFree(d_mat_output1);
        if (err != hipSuccess)
        {
            fprintf(stderr, "Failed to free device vector d_mat_output1 (error code %s)!\n", hipGetErrorString(err));
            exit(EXIT_FAILURE);
        }
        
        print_matrix(h_mat_output1,mat_dim,mat_dim);
        
        free(h_mat_input1);
        free(h_mat_output1);
        free(h_mat_output2);
        free(h_mat_conv_input);

        err = hipDeviceReset();
        if (err != hipSuccess)
        {
            fprintf(stderr, "Failed to deinitialize the device! error=%s\n", hipGetErrorString(err));
            exit(EXIT_FAILURE);
        }

        // printf("Done\n");
    }
    return 0;
}