#include "hip/hip_runtime.h"
%%cuda --name week4_Programming_Assignment_b.cu
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

void print_matrix(float *A, int N,int M){
    for(int i=0;i<N;i++){
        for(int j=0;j<M;j++){
            printf("%.2f ",A[i*N+j]);}
        printf("\n");
    }
}

__global__ void
swap_it(float *A,int M,int N){
    //swapping the even and odd positions

    //calculate the index
    int i=threadIdx.z*(blockDim.x* blockDim.y)+threadIdx.y*blockDim.x+threadIdx.x;

    if(i<M*N){
        if(i%2==1){
            int temp;
            temp=A[i];
            A[i]=A[i-1];
            A[i-1]=temp;
        }

        __syncthreads();
    }
}

__global__ void
diagonal_swap(float *A,int M,int N){
    //taking the transpose
    int i=blockIdx.y*blockDim.y+threadIdx.y;
    int j=blockIdx.x*blockDim.x+threadIdx.x;
    int t=i*N+j;
    int k=j*M+i;
    
    if(i<M && j<N){   
        if(i<j){
          float temp=A[t];
          A[t]=A[k];
          A[k]=temp;
        }
    }
}

int main(void){

    hipError_t err = hipSuccess;

    int t;
    scanf("%d",&t);
    
    while(t--){
        int m,n;
        scanf("%d %d",&m,&n);
        size_t size = m*n * sizeof(float);

        float *h_A=(float *)malloc(size);
        
        // Verify that allocations succeeded
        if (h_A == NULL){
            fprintf(stderr, "Failed to allocate host vectors!\n");
            exit(EXIT_FAILURE);
        }

        for (int i = 0; i < n*m; ++i){
            scanf("%f",&h_A[i]);
        }
        
        // Allocate the device input vector A
        float *d_A = NULL;
        err=hipMalloc (( void **) &d_A , size );

        // Copy the host input vectors A and B in host memory to the device input vectors in device memory
        err = hipMemcpy (d_A , h_A , size , hipMemcpyHostToDevice );
        
        swap_it<<<1, dim3(512,2,1)>>>(d_A, m, n);
        err = hipGetLastError();

        if (err != hipSuccess){
            fprintf(stderr, "Failed to launch vectorAdd kernel (error code %s)!\n", hipGetErrorString(err));
            exit(EXIT_FAILURE);
        }
        err = hipMemcpy (h_A , d_A , size , hipMemcpyDeviceToHost );
//--------------------------------------------------------------------------------------------------------------------------------------
//         print_matrix(h_A,m,n);
//         printf("\n----------------------------\n");
//--------------------------------------------------------------------------------------------------------------------------------------
        diagonal_swap<<<1, dim3(m,n)>>>(d_A, m, n);
        err = hipGetLastError();

        if (err != hipSuccess){
            fprintf(stderr, "Failed to launch vectorAdd kernel (error code %s)!\n", hipGetErrorString(err));
            exit(EXIT_FAILURE);
        }
        // Copy the device result vector in device memory to the host result vector in host memory.
        err = hipMemcpy (h_A , d_A , size , hipMemcpyDeviceToHost );
//--------------------------------------------------------------------------------------------------------------------------------------
         print_matrix(h_A,m,n);
        
        // Free memory
        hipFree(d_A);
        free(h_A);

        err = hipDeviceReset();

        if (err != hipSuccess)
        {
            fprintf(stderr, "Failed to deinitialize the device! error=%s\n", hipGetErrorString(err));
            exit(EXIT_FAILURE);
        }
    }
    return 0;
}